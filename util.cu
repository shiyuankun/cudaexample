#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "matrix.h"

int
CmpMat(Matrix l, Matrix r)
{
    int i, j;
    if (l == NULL || r == NULL)
    {
        printf("NULL pointer\n");
        return 0;
    }
    
    if (l->height != r->height)
        return 0;
    if (l->width != r->width)
        return 0;
    
    for (i = 0;i<l->height;i ++ )
        for (j = 0;j<l->width;j ++ )
            if (fabs(l->elements[i*l->width + j] - r->elements[i*r->width + j])>0.00001)
            {
                return 0;
            }
    return 1;
}

Matrix
Initialize(int width, int height, int initval)
{
    Matrix x = (Matrix)malloc(sizeof(struct MatrixStruct));
    int i, j;
    x->width = width;
    x->height = height;
    x->pitch = width;
    x->elements = (float *)malloc(width * height * sizeof(float));
    if (initval)
    {
        for (i = 0;i < x->height;i++)
            for (j = 0;j < x->width;j++)
                x->elements[i * x->width + j] = i*10.0 + j;
    }
    return x;
}

void
CopyToDeviceMatrix(Matrix Mdevice, const Matrix Mhost)
{
    int size = Mhost->width * Mhost->height * sizeof(float);
    hipMemcpy(Mdevice->elements, Mhost->elements, size, hipMemcpyHostToDevice);
}

void
CopyFromDeviceMatrix(Matrix Mhost, const Matrix Mdevice)
{
    int size = Mdevice->width * Mdevice->height * sizeof(float);
    hipMemcpy(Mhost->elements, Mdevice->elements, size, hipMemcpyDeviceToHost);
}

void
FreeDeviceMatrix(Matrix Mdevice)
{
    hipFree(Mdevice->elements);
    free(Mdevice);
}

void
FreeMatrix(Matrix Mhost)
{
    free(Mhost->elements);
    free(Mhost);
}


Matrix
InitializeDevice(Matrix Mhost)
{
    Matrix Mdevice = (Matrix)malloc(sizeof(struct MatrixStruct));
    Mdevice->width = Mhost->width;
    Mdevice->height = Mhost->height;
    Mdevice->pitch = Mhost->pitch;
    int size = Mdevice->width * Mdevice->height * sizeof(float);
    hipMalloc((void **)&Mdevice->elements, size);
    return Mdevice;
}

Matrix
TransposeHost(const Matrix Mhost)
{
    int i, j;
    Matrix T = Initialize(Mhost->height, Mhost->width, 0);
    
    for (i = 0;i < Mhost->height;i++)
        for (j = 0;j < Mhost->width;j++)
            T->elements[j * T->width + i] = Mhost->elements[i * Mhost->width + j];
    return T;
}

void
PrintMatrix(Matrix Mhost)
{
    int i, j;
    for (i = 0;i < Mhost->height;i++)
        for (j=0;j < Mhost->width;j++)
        {
            printf("%f%s", Mhost->elements[i * Mhost->width + j], j == Mhost->width - 1 ? "\n" : " ");
        }
}

